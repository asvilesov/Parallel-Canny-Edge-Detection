#include "hip/hip_runtime.h"


#include<string>
#include<map>
#include<iostream> 
#include<math.h>
#include<stdlib.h>
#include <hipblas.h> //CUDA
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>


using namespace cv;
using namespace std;

__device__ __managed__ int flag = 1; // 0 if unchanged, 1 if changed in the Hystersis+thresholding function

string type2str(int type) {
  string r;

  uchar depth = type & CV_MAT_DEPTH_MASK;
  uchar chans = 1 + (type >> CV_CN_SHIFT);

  switch ( depth ) {
    case CV_8U:  r = "8U"; break;
    case CV_8S:  r = "8S"; break;
    case CV_16U: r = "16U"; break;
    case CV_16S: r = "16S"; break;
    case CV_32S: r = "32S"; break;
    case CV_32F: r = "32F"; break;
    case CV_64F: r = "64F"; break;
    default:     r = "User"; break;
  }

  r += "C";
  r += (chans+'0');

  return r;
}

__global__ void convolution_kernel(int *img, int *conv, int *phase, int *h, int *w, int *padding){
	int my_x = threadIdx.x;
	int my_y = (blockIdx.x+*padding)*(blockDim.x+2*(*padding));
	int x_gradient[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
	int y_gradient[] = {-1,- 2, -1, 0, 0, 0, +1, +2, +1};

	//int y_gradient[] = {0,0,0, 0, 1, 0, 0, 0, 0};

	int x_mag, y_mag;

	*padding = 1;

	//remove the 0 multiplications... they are useless
	// x_mag = img[my_y*(*padding-1)+my_x+*padding-1]*x_gradient[0] +  img[my_y*(*padding-1)+my_x+*padding]*x_gradient[1] + img[my_y*(*padding-1)+my_x+*padding+1]*x_gradient[2] +  
	// 		img[my_y*(*padding)+my_x+*padding-1]*x_gradient[3] +  img[my_y*(*padding)+my_x+*padding]*x_gradient[4] + img[my_y*(*padding)+my_x+*padding+1]*x_gradient[5] + 
	// 		img[my_y*(*padding+1)+my_x+*padding-1]*x_gradient[6] +  img[my_y*(*padding+1)+my_x+*padding]*x_gradient[7] + img[my_y*(*padding+1)+my_x+*padding+1]*x_gradient[8];
	x_mag = img[(my_y-blockDim.x+2*(*padding))+my_x+*padding-1]*x_gradient[0] +  img[(my_y-blockDim.x+2*(*padding))+my_x+*padding]*x_gradient[1] + img[(my_y-blockDim.x+2*(*padding))+my_x+*padding+1]*x_gradient[2] +  
			img[my_y+my_x+*padding-1]*x_gradient[3] +  img[my_y+my_x+*padding]*x_gradient[4] + img[my_y+my_x+*padding+1]*x_gradient[5] + 
			img[(my_y+blockDim.x+2*(*padding))+my_x+*padding-1]*x_gradient[6] +  img[(my_y+blockDim.x+2*(*padding))+my_x+*padding]*x_gradient[7] + img[(my_y+blockDim.x+2*(*padding))+my_x+*padding+1]*x_gradient[8];

	y_mag = img[(my_y-blockDim.x+2*(*padding))+my_x+*padding-1]*y_gradient[0] +  img[(my_y-blockDim.x+2*(*padding))+my_x+*padding]*y_gradient[1] + img[(my_y-blockDim.x+2*(*padding))+my_x+*padding+1]*y_gradient[2] +  
			img[my_y+my_x+*padding-1]*y_gradient[3] +  img[my_y+my_x+*padding]*y_gradient[4] + img[my_y+my_x+*padding+1]*y_gradient[5] + 
			img[(my_y+blockDim.x+2*(*padding))+my_x+*padding-1]*y_gradient[6] +  img[(my_y+blockDim.x+2*(*padding))+my_x+*padding]*y_gradient[7] + img[(my_y+blockDim.x+2*(*padding))+my_x+*padding+1]*y_gradient[8];

	// if (int(sqrt(float(y_mag*y_mag))) > 255){
	// 	y_mag = 0;
	// }

	conv[(my_y) + my_x + *padding] = int(sqrt(float(y_mag*y_mag)+ float(x_mag*x_mag))/758*255); //CUDA only accpets floats/double in fp operations
	float phase_angle = atan2(float(y_mag), float(x_mag)) * 180 / (atan(1.0)*4);
	if ( phase_angle < 0){
		phase_angle += 180;
	}
	phase[(my_y) + my_x + *padding] = phase_angle;
}

//3x3 Kernel hard-coded with 1-sigma SD
__global__ void gaussian_filter(int *img, int *conv, int *padding){
	int my_x = threadIdx.x;
	int my_y = (blockIdx.x+*padding)*(blockDim.x+2*(*padding));
	
	float gauss[] = {0.077847,	0.123317,	0.077847,
					0.123317,	0.195346,	0.123317,
					0.077847,	0.123317,	0.077847};

	float gauss_val = 	img[(my_y-blockDim.x+2*(*padding))+my_x+*padding-1]*gauss[0] +  img[(my_y-blockDim.x+2*(*padding))+my_x+*padding]*gauss[1] + img[(my_y-blockDim.x+2*(*padding))+my_x+*padding+1]*gauss[2] +  
						img[my_y+my_x+*padding-1]*gauss[3] +  img[my_y+my_x+*padding]*gauss[4] + img[my_y+my_x+*padding+1]*gauss[5] + 
						img[(my_y+blockDim.x+2*(*padding))+my_x+*padding-1]*gauss[6] +  img[(my_y+blockDim.x+2*(*padding))+my_x+*padding]*gauss[7] + img[(my_y+blockDim.x+2*(*padding))+my_x+*padding+1]*gauss[8];

	conv[(my_y) + my_x + *padding] = int(gauss_val); 

}

__global__ void non_max_suppression(int *img, int *output, int *phase, int *padding){ //this might have an error, if it is changing in place... should be reading from steady_state image
	int my_x = threadIdx.x;
	int my_y = (blockIdx.x+*padding)*(blockDim.x+2*(*padding));

	int phase_to_pix[8][4] = {{0,1,0,-1} , {-1,1,1,-1}, {-1,1,1,-1}, {1,0,-1,0}, {1,0,-1,0}, {1,1,-1,-1}, {1,1,-1,-1}, {0,1,0,-1}}; //{(x1,y1),(x2,y2)} for each possible direction

	int compare_value = img[my_y+my_x+*padding];
	float i = phase[my_y+my_x+*padding];
	int val = int(i/22.5);

	if( (compare_value < img[(my_y+blockDim.x*phase_to_pix[val][1])+my_x+*padding + phase_to_pix[val][0]]) || (compare_value < img[(my_y+blockDim.x*phase_to_pix[val][2])+my_x+*padding + phase_to_pix[val][3]])){
		output[my_y+my_x+*padding] = 0;
	}
}

__global__ void thresholding(int *img, int *padding, int *high, int *low){
	int my_x = threadIdx.x;
	int my_y = (blockIdx.x+*padding)*(blockDim.x+2*(*padding));

	if(img[my_y+my_x+*padding] > *high)
		img[my_y+my_x+*padding] = 255;
	else if(img[my_y+my_x+*padding] < *low)
		img[my_y+my_x+*padding] = 0;
	else
		img[my_y+my_x+*padding] = 100;
}

__global__ void hystersis(int*img, int*padding){
	int my_x = threadIdx.x;
	int my_y = (blockIdx.x+*padding)*(blockDim.x+2*(*padding));

	if(img[my_y+my_x+*padding] == 100){
		if( (img[(my_y-blockDim.x+2*(*padding))+my_x+*padding-1] == 255) || (img[(my_y-blockDim.x+2*(*padding))+my_x+*padding]== 255) || 
			(img[(my_y-blockDim.x+2*(*padding))+my_x+*padding+1] == 255) || (img[my_y+my_x+*padding-1] == 255) ||
			(img[(my_y+blockDim.x+2*(*padding))+my_x+*padding-1] == 255) || (img[my_y+my_x+*padding+1] == 255) ||
			(img[(my_y+blockDim.x+2*(*padding))+my_x+*padding] == 255) || (img[(my_y+blockDim.x+2*(*padding))+my_x+*padding+1]== 255)  ){

			img[my_y+my_x+*padding] = 255;
			flag = 1;
		}

		if( (img[(my_y-blockDim.x+2*(*padding))+my_x+*padding-1] == 0) || (img[(my_y-blockDim.x+2*(*padding))+my_x+*padding]== 0) || 
			(img[(my_y-blockDim.x+2*(*padding))+my_x+*padding+1] == 0) || (img[my_y+my_x+*padding-1] == 0) ||
			(img[(my_y+blockDim.x+2*(*padding))+my_x+*padding-1] == 0) || (img[my_y+my_x+*padding+1] == 0) ||
			(img[(my_y+blockDim.x+2*(*padding))+my_x+*padding] == 0) || (img[(my_y+blockDim.x+2*(*padding))+my_x+*padding+1]== 0)  ){

			img[my_y+my_x+*padding] = 0;
			flag = 1;
		}

	}
}




// }

struct pixel_angle{
	int pixel_loc[4];
};


int main(){

	//timing setup
	hipEvent_t start, stop;
	float time_execute = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//image setup
	Mat img_cv = imread("test.png");
	Mat gray_img;
	cvtColor(img_cv, gray_img, CV_BGR2GRAY);
	printf("%s", type2str(gray_img.type()).c_str());

	int height = img_cv.rows;
	int width = img_cv.cols;
	//Canny Edge Filter Parameters
	int high = 50;
	int low = 10;
	int *h_p = &high;
	int *l_p = &low;

	int *img = new int[height*width];
	int *conv_img = new int[height*width];
	int *phase_img = new int[height*width];

	int i = 0;
	for(i = 0; i < height*width; i++){
		img[i] = gray_img.at<uchar>(int(i/width), i%width);
	}

	//Kernel Setup
	int kernel_size = 3;
	int padd = kernel_size/2;
	int *kernel_p = &kernel_size;
	int *padd_p = &padd;
	printf("%i\n", *padd_p);

	//GPU setup
	dim3 dimGrid(height-2*padd);
	dim3 dimBlock(width-2*padd);

	int *gpu_img, *gpu_conv_img, *gpu_phase_img, *gpu_padd, *gpu_h, *gpu_w;
	hipMalloc((void**)&gpu_img, sizeof(int)*height*width);
	hipMalloc((void**)&gpu_conv_img, sizeof(int)*height*width);
	hipMalloc((void**)&gpu_phase_img, sizeof(int)*height*width);
	hipMalloc((void**)&gpu_padd, sizeof(int));
	hipMalloc((void**)&gpu_h, sizeof(int));
	hipMalloc((void**)&gpu_w, sizeof(int));
	// map<int,int*> gpu_angle_pix;
	// map<int,int*> angle_to_pixel_loc;
	// int NS[] {0,1,0,-1};
	// angle_to_pixel_loc[0] = NS;
	// hipMalloc((void**)&gpu_angle_pix, sizeof(angle_to_pixel_loc));


	hipMemcpy(gpu_img, img, sizeof(int)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(gpu_conv_img, conv_img, sizeof(int)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(gpu_phase_img, phase_img, sizeof(int)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(gpu_padd, padd_p, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_h, h_p, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_w, l_p, sizeof(int), hipMemcpyHostToDevice);

	//invoke Gauss Kernel
	gaussian_filter<<<dimGrid, dimBlock>>> (gpu_img, gpu_conv_img, gpu_padd);
	hipMemcpy(conv_img, gpu_conv_img, sizeof(int)*height*width, hipMemcpyDeviceToHost);

	// //invoke Sobel Kernel
	convolution_kernel<<<dimGrid, dimBlock>>> (gpu_conv_img, gpu_img, gpu_phase_img, gpu_h, gpu_w, gpu_padd);
	hipMemcpy(conv_img, gpu_img, sizeof(int)*height*width, hipMemcpyDeviceToHost);
	hipMemcpy(phase_img, gpu_phase_img, sizeof(int)*height*width, hipMemcpyDeviceToHost);

	// // // invoke non-max suppression
	int *gpu_new_img;
	hipMalloc((void**)&gpu_new_img, sizeof(int)*height*width);
	hipMemcpy(gpu_new_img, conv_img, sizeof(int)*height*width, hipMemcpyHostToDevice);
	non_max_suppression<<<dimGrid, dimBlock>>> (gpu_img, gpu_new_img, gpu_phase_img, gpu_padd);
	hipMemcpy(conv_img, gpu_new_img, sizeof(int)*height*width, hipMemcpyDeviceToHost);

	// // // invoke thresholding
	thresholding<<<dimGrid, dimBlock>>> (gpu_new_img, gpu_padd, gpu_h, gpu_w);
	hipMemcpy(conv_img, gpu_new_img, sizeof(int)*height*width, hipMemcpyDeviceToHost);

	// invoke hysteresis
	int count = 0;
	while(flag == 1){
		count++;
		flag = 0;
		hystersis<<<dimGrid, dimBlock>>> (gpu_new_img, gpu_padd);
		hipDeviceSynchronize();
		hipMemcpy(conv_img, gpu_new_img, sizeof(int)*height*width, hipMemcpyDeviceToHost);
	}

	printf("Flag is: %i and count is %i\n", flag, count);


	printf("%i\n", phase_img[1000]);






	//find max element
	printf("%i\n", *max_element(conv_img, conv_img + height*width));


	//printf("%i\n", conv_img[1]);


	namedWindow( "Display window", WINDOW_AUTOSIZE );// Create a window for display.
    imshow( "Display window", gray_img);                   // Show our image inside it.

    //Mat conv_img_cv = Mat(1, width, CV_8U, conv_img, sizeof(int)*width);
    //memcpy(conv_img_cv.data, conv_img, height*width*sizeof(int));
    for(i = 0; i < height*width; i++){
		gray_img.at<uchar>(int(i/width), i%width) = conv_img[i];
	}
    namedWindow( "Convolution Image", WINDOW_AUTOSIZE);
    imshow("Convolution Image", gray_img);

    //printf("%s", type2str(conv_img.type()).c_str());

    waitKey(0);                                          // Wait for a keystroke in the window
	
    printf("Image size: %i %i \n", height, width);
	printf("%i", img_cv.at<Vec3b>(0,1)[1]);

	hipError_t error = hipGetLastError();
	printf("error: %s\n", hipGetErrorString(error));


	delete[] img;
	delete[] conv_img;
	delete[] phase_img;
	hipFree(gpu_img);
	hipFree(gpu_conv_img);
	hipFree(gpu_padd);
	hipFree(gpu_h);
	hipFree(gpu_w);
	hipFree(gpu_phase_img);
	// hipFree(gpu_angle_pix);

	return 0;
}