#include "hip/hip_runtime.h"


#include<string>
#include<iostream> 
#include<stdlib.h>
#include <hipblas.h> //CUDA
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace cv;

__global__ void convolution_kernel(int *img, int *conv, int *h, int *w, int *padding){
	int my_x = threadIdx.x;
	int my_y = blockIdx.x*blockDim.x;
	int x_gradient[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};

	conv[my_y*(1+*padding) + my_x + *padding] = img[my_y*(*padding-1)+my_x+*padding-1]*x_gradient[0] +  img[my_y*(*padding-1)+my_x+*padding]*x_gradient[1] + img[my_y*(*padding-1)+my_x+*padding+1]*x_gradient[2] +  
												img[my_y*(*padding)+my_x+*padding-1]*x_gradient[3] +  img[my_y*(*padding)+my_x+*padding]*x_gradient[4] + img[my_y*(*padding)+my_x+*padding+1]*x_gradient[5] + 
												img[my_y*(*padding+1)+my_x+*padding-1]*x_gradient[6] +  img[my_y*(*padding+1)+my_x+*padding]*x_gradient[7] + img[my_y*(*padding+1)+my_x+*padding+1]*x_gradient[8]; 

}



int main(){

	//timing setup
	hipEvent_t start, stop;
	float time_execute = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//image setup
	Mat img_cv = imread("test.png");
	Mat gray_img;
	cvtColor(img_cv, gray_img, CV_BGR2GRAY);

	int height = img_cv.rows;
	int width = img_cv.cols;
	int *h_p = &height;
	int *w_p = &width;

	int *img = new int[height*width];
	int *conv_img = new int[height*width];

	int i = 0;
	for(i = 0; i < height*width; i++){
		img[i] = gray_img.at<uchar>(int(i/width), i%width);
	}

	printf("%i\n", conv_img[600]);

	//Kernel Setup
	int kernel_size = 3;
	int padd = kernel_size/2;
	int *kernel_p = &kernel_size;
	int *padd_p = &padd;

	//GPU setup
	dim3 dimGrid(height-2*padd);
	dim3 dimBlock(width-2*padd);

	int *gpu_img, *gpu_conv_img, *gpu_padd, *gpu_h, *gpu_w;
	hipMalloc((void**)&gpu_img, sizeof(int)*height*width);
	hipMalloc((void**)&gpu_conv_img, sizeof(int)*height*width);
	hipMalloc((void**)&gpu_padd, sizeof(int));
	hipMalloc((void**)&gpu_h, sizeof(int));
	hipMalloc((void**)&gpu_w, sizeof(int));

	hipMemcpy(gpu_img, img, sizeof(int)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(gpu_conv_img, conv_img, sizeof(int)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(gpu_padd, padd_p, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_h, h_p, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_w, w_p, sizeof(int), hipMemcpyHostToDevice);

	//invoke Kernel
	convolution_kernel<<<dimGrid, dimBlock>>> (gpu_img, gpu_conv_img, gpu_h, gpu_w, gpu_padd);
	hipMemcpy(conv_img, gpu_conv_img, sizeof(int)*height*width, hipMemcpyDeviceToHost);


	printf("%i\n", conv_img[600]);


	namedWindow( "Display window", WINDOW_AUTOSIZE );// Create a window for display.
    imshow( "Display window", gray_img);                   // Show our image inside it.

    waitKey(0);                                          // Wait for a keystroke in the window
	
    printf("Image size: %i %i \n", height, width);
	printf("%i", img_cv.at<Vec3b>(1,1)[1]);

	hipError_t error = hipGetLastError();
	printf("error: %s\n", hipGetErrorString(error));


	delete[] img;
	delete[] conv_img;
	hipFree(gpu_img);
	hipFree(gpu_conv_img);
	hipFree(gpu_padd);
	hipFree(gpu_h);
	hipFree(gpu_w);

	return 0;
}