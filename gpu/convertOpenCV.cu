
#include<string>
#include<iostream> 
#include<stdlib.h>
#include <hipblas.h> //CUDA
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "canny_algorithm.h"

using namespace std;
using namespace cv;

//Should take the folder path of the images and of the folder to save the grayscale photos into
void convertToOpenCV(String folderpath, string saveFolder) {		//void for now, idk what it should return or if we will just pass the data by calling another function

    folderpath = "../images/512x512/*.jpg";  // This is temporary, whatever calls the function should give the folder
                                            // That allows us to test on different folders and not hardcode it
    vector<String> filenames;
    cv::glob(folderpath, filenames);

    for (size_t i = 0; i < filenames.size(); i++)
    {
        //Read image in to program in color
        Mat im = imread(filenames[i]);//, 1;

        //Matrix to hold the images
        Mat grayscaleImage;
        //Convert to grayscale
        cvtColor(im, grayscaleImage, CV_BGR2GRAY);

        String save = saveFolder + "/" + filenames[i] + "_grey";

        //Can run function on the image here
        //printf("Total time of execution: %f\n", canny_edge_detector(gray_img));

        imwrite(save, grayscaleImage);
    }

}

int main(){

}